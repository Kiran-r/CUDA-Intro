
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void MyKernel()
{
    printf("blockIdx.x=%u,ThreadIdx.x=%u\n",blockIdx.x,threadIdx.x);
    return;
}

int main()
{
    printf("Kernel (Blocks x Threads)\n");
    MyKernel<<<1, 2>>>();
    printf("\n\n****Kernel (1x2) launched****\n\n");
    hipDeviceSynchronize();
    printf("\n****Kernel finished****\n\n");
    MyKernel<<<2, 1>>>();
    printf("\n\n****Kernel (2x1) launched****\n\n");
    hipDeviceSynchronize();
    printf("\n****Kernel finished****\n\n");
    MyKernel<<<2, 2>>>();
    printf("\n\n****Kernel (2x2) launched****\n\n");
    hipDeviceSynchronize();
    printf("\n****Kernel finished****\n\n");
    return 0;
}
