
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void MyKernel()
{
    printf("ThreadId(x,y,z)=(%u,%u,%u)blockId(x,y,z)=(%u,%u,%u)\n",
            threadIdx.x, threadIdx.y, threadIdx.z,
            blockIdx.x, blockIdx.y, blockIdx.z);
    return;
}

int main()
{
    MyKernel<<<2,2>>>();
    printf("\n\n****Kernel (2x2)=4 threads launched****\n\n");
    hipDeviceSynchronize();
    printf("\n****Kernel finished****\n\n\n\n");

    dim3 numBlks(2,2,2);
    dim3 threadsPerBlk(2,2,2);

    MyKernel<<<numBlks,threadsPerBlk>>>();
    printf("\n\n****Kernel (blocks(2x2x2), threads(2,2,2))=64 threads launched****\n\n");
    hipDeviceSynchronize();
    printf("\n****Kernel finished****\n\n");
    return 0;
}
