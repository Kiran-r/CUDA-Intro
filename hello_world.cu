#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void MyKernel()
{
    printf("threadId[%u]=Hello World\n",threadIdx.x);
    return;
}

int main()
{
    MyKernel<<<1,1>>>();
    printf("****Kernel launched****\n\n");
    hipDeviceSynchronize();
    printf("\n****Kernel finished****\n");
    return 0;
}
